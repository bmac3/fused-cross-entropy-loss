#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>
#include <hip/hip_cooperative_groups.h>
#include <stdexcept>
#include "cutlass/bfloat16.h"
#include "cutlass/cutlass.h"
#include "cutlass/fast_math.h"
#include "cutlass/functional.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"
#include "kernels.h"
#include "kernel_helpers.h"
#include "utils.h"

namespace cu_ext {

using namespace cute;

namespace {

template <class T>
CUTE_HOST_DEVICE 
T infinity() {
    return std::numeric_limits<T>::infinity();
}


template <class T, class TC, class CtaTiler, class ReduceTiler,
          class XGmemLayout, class XSmemLayout, class TiledCopyX, 
          class VGmemLayout, class VSmemLayout, class TiledCopyV,
          class YGmemLayout, class YSmemLayout,
          class OGmemLayout, 
          class CSmemLayout, class NSmemLayout, class MSmemLayout,
          class TiledMma>
__global__ void cuFusedCrossEntropyLossFwd(CtaTiler cta_tiler, ReduceTiler reduce_tiler,
                                           T const* X, XGmemLayout mX_layout, XSmemLayout sX_layout, TiledCopyX copy_x,
                                           T const* V, VGmemLayout mV_layout, VSmemLayout sV_layout, TiledCopyV copy_v,
                                           int const* Y, YGmemLayout mY_layout, YSmemLayout sY_layout,
                                           T      * O, OGmemLayout mO_layout,
                                           CSmemLayout sC_layout, NSmemLayout sN_layout, MSmemLayout sM_layout,
                                           TiledMma mma) {
    // get full tensors
    Tensor mX = make_tensor(make_gmem_ptr(X), mX_layout);   // (b,e)
    Tensor mV = make_tensor(make_gmem_ptr(V), mV_layout);   // (v,e)
    Tensor mY = make_tensor(make_gmem_ptr(Y), mY_layout);   // (b)
    Tensor mO = make_tensor(make_gmem_ptr(O), mO_layout);   // (b)

    // get appropriate blocks for this thread block
    auto cta_coord = make_coord(blockIdx.x, _, _);
    Tensor gX = local_tile(mX, cta_tiler, cta_coord, Step<_1,cute::X,_1>{});          // (BLK_B,BLK_E,e)
    Tensor gV = flat_divide(mV, select<1,2>(cta_tiler));                              // (BLK_V,BLK_E,v,e)
    Tensor gY = local_tile(mY, cta_tiler, cta_coord, Step<_1,cute::X,cute::X>{});     // (BLK_B)
    Tensor gO = local_tile(mO, cta_tiler, cta_coord, Step<_1,cute::X,cute::X>{});     // (BLK_B)

    // create shared memory buffers for local/online normalizers, max scores, and label scores
    __shared__ float smemLN[cosize_v<NSmemLayout>]; 
    __shared__ float smemLM[cosize_v<MSmemLayout>];
    __shared__ float smemON[cosize_v<NSmemLayout>]; 
    __shared__ float smemOM[cosize_v<MSmemLayout>];
    __shared__ T smemX[cosize_v<XSmemLayout>];
    __shared__ T smemV[cosize_v<VSmemLayout>];
    __shared__ TC smemC[cosize_v<CSmemLayout>];

    Tensor sLN = make_tensor(make_smem_ptr(smemLN), sN_layout);     // (BLK_B)
    Tensor sLM = make_tensor(make_smem_ptr(smemLM), sM_layout);     // (BLK_B)
    Tensor sON = make_tensor(make_smem_ptr(smemON), sN_layout);     // (BLK_B)
    Tensor sOM = make_tensor(make_smem_ptr(smemOM), sM_layout);     // (BLK_B)

    Tensor sX = make_tensor(make_smem_ptr(smemX), sX_layout);       // (BLK_B,BLK_E)
    Tensor sV = make_tensor(make_smem_ptr(smemV), sV_layout);       // (BLK_V,BLK_E)
    Tensor sC = make_tensor(make_smem_ptr(smemC), sC_layout);       // (BLK_B,BLK_V)

    // partition X and V for copying
    ThrCopy thr_copy_x = copy_x.get_slice(threadIdx.x);
    Tensor tXgX = thr_copy_x.partition_S(gX);                       // (CPY,CPY_B,CPY_E,e)
    Tensor tXsX = thr_copy_x.partition_D(sX);                       // (CPY,CPY_B,CPY_E)

    ThrCopy thr_copy_v = copy_v.get_slice(threadIdx.x);
    Tensor tVgV = thr_copy_v.partition_S(gV);                       // (CPY,CPY_V,CPY_E,v,e)
    Tensor tVsV = thr_copy_v.partition_D(sV);                       // (CPY,CPY_V,CPY_E)

    // partition X and V for mma
    ThrMMA thr_mma = mma.get_slice(threadIdx.x);
    Tensor tCsX = thr_mma.partition_A(sX);                          // (MMA,MMA_B,MMA_E)
    Tensor tCsV = thr_mma.partition_B(sV);                          // (MMA,MMA_V,MMA_E)
    Tensor tCsC = thr_mma.partition_C(sC);                          // (MMA,MMA_B,MMA_V)

    // create accumulator in registers
    Tensor tCrC = thr_mma.make_fragment_C(tCsC);                    // (MMA,MMA_B,MMA_V)

    auto V_BLOCK_MAX = size<3>(tVgV);
    auto V_BLOCK_SIZE = size<0>(sV);
    auto E_TILE_MAX = size<3>(tXgX);
    auto B_REG_MAX = size<1>(tCrC);
    auto V_REG_MAX = size<2>(tCrC);

    // register vars for local stats
    float r_sum;
    float r_max;
    
    // register vars for label logits
    CUTE_STATIC_ASSERT_V(size(mma) >= size(gY));
    int y = (threadIdx.x < size(gY)) ? gY(threadIdx.x) : -1;
    TC label_logit;

    // partition C for reduction
    auto reduce_coord = idx2crd(threadIdx.x, shape(reduce_tiler));
    Tensor tRsC = local_partition(sC, reduce_tiler, threadIdx.x);

    // partition local stat arrays for reduction
    Tensor tRsLM = local_partition(sLM, get<0>(reduce_tiler), get<0>(reduce_coord));
    Tensor tRsLN = local_partition(sLN, get<0>(reduce_tiler), get<0>(reduce_coord));

    // init online stats
    fill(sOM, -infinity<float>());
    fill(sON, 0.f);

    // TODO: deal with imperfect tiling
    for (int v_block = 0; v_block < V_BLOCK_MAX; ++v_block) {
        // clear the accumulator
        clear(tCrC);
        for (int e_tile = 0; e_tile < E_TILE_MAX; ++e_tile) {
            // copy X and V from gmem to smem
            copy(copy_x, tXgX(_,_,_,e_tile),         tXsX);
            copy(copy_v, tVgV(_,_,_,v_block,e_tile), tVsV);
            __syncthreads();

            // compute gemm
            gemm(mma, tCsX, tCsV, tCrC);
            __syncthreads();
        }
        // copy gemm tile to smem
        copy(tCrC, tCsC);
        __syncthreads();

        // initialize local/register normalizer and max values
        fill(sLM, -infinity<float>());
        r_max = -infinity<float>();
        fill(sLN, 0.f);
        r_sum = 0.f;
        __syncthreads();

        // take label logits
        if ((y >= v_block * V_BLOCK_SIZE) && (y < (v_block+1) * V_BLOCK_SIZE)) {
            label_logit = sC(threadIdx.x, y - v_block * V_BLOCK_SIZE);
        }

        // compute register max
        for (int i = 0; i < size(tRsC); ++i) {
            r_max = max(r_max, static_cast<float>(tRsC(i)));
        }

        // compute local max
        cutlass::atomic_maximum<float>{}(&tRsLM(0), r_max);
        __syncthreads();

        // compute register sum
        r_max = tRsLM(0);
        for (int i = 0; i < size(tRsC); ++i) {
            r_sum += exp(tRsC(i) - r_max);
        }

        // compute local sum
        cutlass::atomic_add<float>{}(&tRsLN(0), r_sum);
        __syncthreads();

        // update online stats
        if (threadIdx.x < size(sON)) {
            r_max = max(sLM(threadIdx.x), sOM(threadIdx.x));
            
            // update online normalizer
            float local_term = (sLN(threadIdx.x) == 0) ? 0 : exp(sLM(threadIdx.x) - r_max) * sLN(threadIdx.x);
            float online_term = (sON(threadIdx.x) == 0) ? 0 : exp(sOM(threadIdx.x) -  r_max) * sON(threadIdx.x);
            sON(threadIdx.x) = local_term + online_term;

            // update online max
            sOM(threadIdx.x) = r_max;
        }
        __syncthreads();
    }

    // save result
    if (threadIdx.x < size(gY)) {
        gO(threadIdx.x) = static_cast<T>(log(sON(threadIdx.x)) - label_logit + sOM(threadIdx.x));
    }

}


void ThrowIfError(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}


template <class T, class TC>
void apply_fused_ce_loss_fwd(hipStream_t stream, void **buffers, const char *opaque, 
                             std::size_t opaque_len) {

    const FusedCELossDescriptor &d = *UnpackDescriptor<FusedCELossDescriptor>(opaque, opaque_len);
    // we can treat batch size and seq len as same dimension since no tokens depend on any other token for
    // this op
    const int batch_size = d.batch_size * d.sequence_len;
    const int vocab_size = d.vocab_size;
    const int embed_size = d.embed_size;

    const T *X = static_cast<const T*>(buffers[0]);
    const T *V = static_cast<const T*>(buffers[1]);
    const int *Y = static_cast<const int*>(buffers[2]);
    T *O = static_cast<T*>(buffers[3]);

    // define gmem layouts
    auto mX = make_layout(make_shape(batch_size, embed_size), LayoutRight{});
    auto mV = make_layout(make_shape(vocab_size, embed_size), LayoutRight{});
    auto mY = make_layout(batch_size);
    auto mO = make_layout(batch_size);

    // define CTA tile sizes
    auto bB = Int<64>{};
    auto bV = Int<64>{};
    auto bE = Int<  8>{};
    auto cta_tiler = make_shape(bB, bV, bE);

    // define smem layouts
    auto sX = make_layout(make_shape(bB, bE), LayoutRight{});
    auto sV = make_layout(make_shape(bV, bE), LayoutRight{});
    auto sY = make_layout(bB);
    auto sC = make_layout(make_shape(bB, bV), LayoutRight{});
    auto sN = make_layout(bB);
    auto sM = make_layout(bB);

    // define reduction tiler
    auto reduce_tiler = make_layout(make_shape(Int<64>{}, Int<4>{}));

    TiledCopy copyX = make_tiled_copy(Copy_Atom<UniversalCopy<T>, T>{},
                                      Layout<Shape<_32,_8>,Stride<_8,_1>>{},
                                      Layout<Shape<_1,_1>>{});

    TiledCopy copyV = make_tiled_copy(Copy_Atom<UniversalCopy<T>, T>{},
                                      Layout<Shape<_32,_8>,Stride<_8,_1>>{},
                                      Layout<Shape<_1,_1>>{});

    TiledMMA mma = make_tiled_mma(UniversalFMA<TC,T,T>{},
                                  Layout<Shape<_16,_16,_1>>{});

    dim3 dimBlock(size(mma));
    dim3 dimGrid(size(ceil_div(batch_size, bB)));

    cuFusedCrossEntropyLossFwd<T, TC><<<dimGrid, dimBlock>>>
        (cta_tiler, reduce_tiler,
         X, mX, sX, copyX,
         V, mV, sV, copyV,
         Y, mY, sY,
         O, mO,
         sC, sN, sM,
         mma);

    ThrowIfError(hipGetLastError());
}

}

void fused_ce_loss_fwd_bf16(hipStream_t stream, void **buffers, const char *opaque,
                       std::size_t opaque_len) {
    apply_fused_ce_loss_fwd<cutlass::bfloat16_t, float>(stream, buffers, opaque, opaque_len);
}

}
