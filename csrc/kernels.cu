#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void add_kernel(int a, int b, int *c) {
    *c = a + b + 12;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    add_kernel<<<1, 1>>>(a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;
}
